#include <iostream>
#include <hip/hip_runtime.h>

class Perf
{
public:
    Perf(const std::string &name)
    {
        m_name = name;
        hipEventCreate(&m_start);
        hipEventCreate(&m_end);
        hipEventRecord(m_start);
        hipEventSynchronize(m_start);
    }

    ~Perf()
    {
        hipEventRecord(m_end);
        hipEventSynchronize(m_end);
        float elapsed_time = 0.0;
        hipEventElapsedTime(&elapsed_time, m_start, m_end);
        std::cout << m_name << " elapse: " << elapsed_time << " ms" << std::endl;
    }

private:
    std::string m_name;
    hipEvent_t m_start, m_end;
}; // class Perf

bool check(float *cpu_result, float *gpu_result, const int M, const int N)
{
    const int size = M * N;
    for (int i = 0; i < size; i++)
    {
        if (cpu_result[i] != gpu_result[i])
        {
            return false;
        }
    }
    return true;
}

__global__ void transpose_naive(float *input, float *output, const int M, const int N)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * N + x;
    int trans_idx = x * M + y;
    output[trans_idx] = input[idx];
}

void transpose_cpu(float *input, float *output, const int M, const int N)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            const int input_index = m * N + n;
            const int output_index = n * M + m;
            output[output_index] = input[input_index];
        }
    }
}

int main(int argc, char *argv[])
{
    const int MATRIX_M = 2048;
    const int MATRIX_N = 512;
    const size_t size = MATRIX_M * MATRIX_N;

    float *input_host = (float *)malloc(size * sizeof(float));
    float *output_host_cpu_calc = (float *)malloc(size * sizeof(float));
    float *output_host_gpu_calc = (float *)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++)
    {
        input_host[i] = 2.0 * (float)drand48() - 1.0;
    }

    transpose_cpu(input_host, output_host_cpu_calc, MATRIX_M, MATRIX_N);
    float *input_device, *output_device;

    hipMalloc(&input_device, size * sizeof(float));
    hipMemcpy(input_device, input_host, size * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&output_device, size * sizeof(float));

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_32_8");
        dim3 block_size(32, 8);
        dim3 grid_size((MATRIX_N - 1) / block_size.x + 1, (MATRIX_M - 1) / block_size.y + 1);
        transpose_naive<<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_16_16");
        dim3 block_size(16, 16);
        dim3 grid_size((MATRIX_N - 1) / block_size.x + 1, (MATRIX_M - 1) / block_size.y + 1);
        transpose_naive<<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_8_32");
        dim3 block_size(8, 32);
        dim3 grid_size((MATRIX_N - 1) / block_size.x + 1, (MATRIX_M - 1) / block_size.y + 1);
        transpose_naive<<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }

    return 0;
}