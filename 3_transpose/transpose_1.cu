//ncu --set full -o /home/yds/3_transpose/transpose_1_report ./3_transpose/transpose_1

#include <iostream>
#include <hip/hip_runtime.h>

class Perf
{
public:
    Perf(const std::string &name)
    {
        m_name = name;
        hipEventCreate(&m_start);
        hipEventCreate(&m_end);
        hipEventRecord(m_start);
        hipEventSynchronize(m_start);
    }

    ~Perf()
    {
        hipEventRecord(m_end);
        hipEventSynchronize(m_end);
        float elapsed_time = 0.0;
        hipEventElapsedTime(&elapsed_time, m_start, m_end);
        std::cout << m_name << " elapse: " << elapsed_time << " ms" << std::endl;
    }

private:
    std::string m_name;
    hipEvent_t m_start, m_end;
}; // class Perf

bool check(float *cpu_result, float *gpu_result, const int M, const int N)
{
    const int size = M * N;
    for (int i = 0; i < size; i++)
    {
        if (cpu_result[i] != gpu_result[i])
        {
            return false;
        }
    }
    return true;
}

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
template <
    const int THREAD_SIZE_Y, // height of block that each thread calculate
    const int THREAD_SIZE_X> // width of block that each thread calculate
__global__ void transpose_float4_inner_4x4(float *input,
                                           float *output, const int M, const int N)
{
    float src_transpose[4][4];
    float dst_transpose[4][4];
    float *input_start = input + N * blockIdx.y * THREAD_SIZE_Y + blockIdx.x * THREAD_SIZE_X;
    for (int i = 0; i < 4; i++)
    {
        FETCH_FLOAT4(src_transpose[i]) = FETCH_FLOAT4(input_start[(threadIdx.y * 4 + i) * N + threadIdx.x * 4]);
    }

    FETCH_FLOAT4(dst_transpose[0]) = make_float4(src_transpose[0][0], src_transpose[1][0], src_transpose[2][0], src_transpose[3][0]);
    FETCH_FLOAT4(dst_transpose[1]) = make_float4(src_transpose[0][1], src_transpose[1][1], src_transpose[2][1], src_transpose[3][1]);
    FETCH_FLOAT4(dst_transpose[2]) = make_float4(src_transpose[0][2], src_transpose[1][2], src_transpose[2][2], src_transpose[3][2]);
    FETCH_FLOAT4(dst_transpose[3]) = make_float4(src_transpose[0][3], src_transpose[1][3], src_transpose[2][3], src_transpose[3][3]);

    float *output_start = output + M * blockIdx.x * THREAD_SIZE_X + blockIdx.y * THREAD_SIZE_Y;
    for (int i = 0; i < 4; i++)
    {
        FETCH_FLOAT4(output_start[(i + threadIdx.x * 4) * M + threadIdx.y * 4]) = FETCH_FLOAT4(dst_transpose[i][0]);
    }
}

void transpose_cpu(float *input, float *output, const int M, const int N)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            const int input_index = m * N + n;
            const int output_index = n * M + m;
            output[output_index] = input[input_index];
        }
    }
}

int main(int argc, char *argv[])
{
    const int MATRIX_M = 2048;
    const int MATRIX_N = 512;
    const size_t size = MATRIX_M * MATRIX_N;

    float *input_host = (float *)malloc(size * sizeof(float));
    float *output_host_cpu_calc = (float *)malloc(size * sizeof(float));
    float *output_host_gpu_calc = (float *)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++)
    {
        input_host[i] = 2.0 * (float)drand48() - 1.0;
    }

    transpose_cpu(input_host, output_host_cpu_calc, MATRIX_M, MATRIX_N);
    float *input_device, *output_device;

    hipMalloc(&input_device, size * sizeof(float));
    hipMemcpy(input_device, input_host, size * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&output_device, size * sizeof(float));

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_32_8");
        dim3 block_size(32, 8);
        dim3 grid_size(((MATRIX_N >> 2) - 1) / block_size.x + 1, ((MATRIX_M >> 2) - 1) / block_size.y + 1);
        constexpr int THREAD_SIZE_Y = 8 * 4;
        constexpr int THREAD_SIZE_X = 32 * 4;
        transpose_float4_inner_4x4<THREAD_SIZE_Y, THREAD_SIZE_X><<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_16_16");
        dim3 block_size(16, 16);
        dim3 grid_size(((MATRIX_N >> 2) - 1) / block_size.x + 1, ((MATRIX_M >> 2) - 1) / block_size.y + 1);
        constexpr int THREAD_SIZE_Y = 16 * 4;
        constexpr int THREAD_SIZE_X = 16 * 4;
        transpose_float4_inner_4x4<THREAD_SIZE_Y, THREAD_SIZE_X><<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }
    else std::cout << "wrong!" << std::endl;

    // ==================
    hipMemset(output_device, 0, size * sizeof(float));
    for (int i = 0; i < 5; i++)
    {
        Perf perf("transpose_8_32");
        dim3 block_size(8, 32);
        dim3 grid_size(((MATRIX_N >> 2) - 1) / block_size.x + 1, ((MATRIX_M >> 2) - 1) / block_size.y + 1);
        constexpr int THREAD_SIZE_Y = 32 * 4;
        constexpr int THREAD_SIZE_X = 8 * 4;
        transpose_float4_inner_4x4<THREAD_SIZE_Y, THREAD_SIZE_X><<<grid_size, block_size>>>(input_device, output_device, MATRIX_M, MATRIX_N);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_host_gpu_calc, output_device,
               size * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output_host_cpu_calc, output_host_gpu_calc, MATRIX_M, MATRIX_N))
    {
        std::cout << "right!" << std::endl;
    }

    return 0;
}