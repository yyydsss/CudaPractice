
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>

#define A(i, j) a[(i) * n + (j)]
#define B(i, j) b[(i) * n + (j)]

#define checkCudaError(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

void random_matrix(int m, int n, float *a)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
#if 1
            A(i, j) = 2.0 * (float)drand48() - 1.0;
#else
            A(i, j) = (j - i) % 3;
#endif
}

float compare_matrices(int m, int n, float *a, float *b)
{
    int i, j;
    float max_diff = 0.0, diff;
    int printed = 0;

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            diff = fabs(A(i, j) - B(i, j));
            max_diff = (diff > max_diff ? diff : max_diff);
            if (0 == printed)
                if (max_diff > 0.5f || max_diff < -0.5f)
                {
                    printf("\n error: i %d  j %d diff %f  got %f  expect %f ", i, j, max_diff, A(i, j), B(i, j));
                    printed = 1;
                }
        }
    }
    return max_diff;
}

void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            float temp = 0.f;
            for (int k = 0; k < K; k++)
            {
                temp += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
            C_ptr[m * N + n] = temp;
        }
    }
}

__global__ void cuda_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    float *A_ptr_start = A_ptr + blockDim.y * blockIdx.y * K;
    float *B_ptr_start = B_ptr + blockDim.x * blockIdx.x;
    float temp = 0.f;
    for (int k = 0; k < K; k++)
    {
        temp += A_ptr_start[threadIdx.y * K + k] * B_ptr_start[k * N + threadIdx.x];
    }
    C_ptr[x + y * N] = temp;
}

int main()
{
    int m = 512;
    int n = 512;
    int k = 512;
    const size_t mem_size_A = m * k * sizeof(float);
    const size_t mem_size_B = k * n * sizeof(float);
    const size_t mem_size_C = m * n * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    random_matrix(m, k, matrix_A_host);
    random_matrix(k, n, matrix_B_host);
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    checkCudaError(hipMalloc((void **)&matrix_A_device, mem_size_A));
    checkCudaError(hipMalloc((void **)&matrix_B_device, mem_size_B));
    checkCudaError(hipMalloc((void **)&matrix_C_device, mem_size_C));

    checkCudaError(hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice));

    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

    constexpr int BLOCK = 8;
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);
    cuda_sgemm<<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, m, n, k);

    // Check for kernel launch errors
    checkCudaError(hipGetLastError());

    // Check for kernel execution errors
    checkCudaError(hipDeviceSynchronize());

    checkCudaError(hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost));

    float diff = compare_matrices(m, n, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc);
    if (diff > 0.5f || diff < -0.5f)
    {
        printf("diff too big !\n");
        exit(-1);
    }
    else
    {
        printf("right\n");
    }

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    checkCudaError(hipFree(matrix_A_device));
    checkCudaError(hipFree(matrix_B_device));
    checkCudaError(hipFree(matrix_C_device));
    return 0;
}